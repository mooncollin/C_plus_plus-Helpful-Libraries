#include <cstddef>
#include <utility>
#include <cmath>

#include <hip/hip_runtime.h>

namespace cmoon::cuda
{
	template<class T>
	__global__
	void matrix_multiply_kernel(int m, int n, int k, const T* const input1, const T* const input2, T* out) noexcept
	{
		const auto row {blockIdx.y * blockDim.y + threadIdx.y};
		const auto column {blockIdx.x * blockDim.x + threadIdx.x};

		if (row < m && column < k)
		{
			for (int i {0}; i < n; ++i)
			{
				out[row * k + column] += input1[row * n + i] * input2[i * k + column];
			}
		}
	}

	template<class T>
	__global__
	void square_matrix_multiply_kernel(int n, const T* const input1, const T* const input2, T* out) noexcept
	{
		constexpr unsigned int block_size {16};
		__shared__ int tile_a[block_size][block_size];
		__shared__ int tile_b[block_size][block_size];

		const auto row {blockIdx.y * block_size + threadIdx.y};
		const auto column {blockIdx.x * block_size + threadIdx.x};
		int tmp {0};

		for (int sub {0}; sub < gridDim.x; ++sub)
		{
			auto idx {row * n + sub * block_size + threadIdx.x};
			if (idx >= n * n)
			{
				tile_a[threadIdx.y][threadIdx.x] = 0;
			}
			else
			{
				tile_a[threadIdx.y][threadIdx.x] = input1[idx];
			}

			idx = (sub * block_size + threadIdx.y) * n + column;
			if (idx >= n * n)
			{
				tile_b[threadIdx.y][threadIdx.x] = 0;
			}
			else
			{
				tile_b[threadIdx.y][threadIdx.x] = input2[idx];
			}

			__syncthreads();

			for (int k {0}; k < block_size; ++k)
			{
				tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
			}

			__syncthreads();
		}

		if (row < n && column < n)
		{
			out[row * n + column] = tmp;
		}
	}

	template<class T>
	void matrix_multiply_impl(int m, int n, int k, const T* const input1, const T* const input2, T* out) noexcept
	{
		constexpr unsigned int block_size {16};
		const auto grid_rows {(m + block_size - 1) / block_size};
		const auto grid_cols {(k + block_size - 1) / block_size};

		const ::dim3 dim_grid {grid_cols, grid_rows};
		const ::dim3 dim_block {block_size, block_size};

		if (m == n && n == k)
		{
			square_matrix_multiply_kernel<<<dim_grid, dim_block>>>(n, input1, input2, out);
		}
		else
		{
			matrix_multiply_kernel<<<dim_grid, dim_block>>>(m, n, k, input1, input2, out);
		}
	}

	namespace details
	{
		void dummy_template_instantiator()
		{
			matrix_multiply_impl<float>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<double>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<char>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<unsigned char>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<short>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<unsigned short>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<int>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<unsigned int>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<long>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<unsigned long>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<long long>(0, 0, 0, nullptr, nullptr, nullptr);
			matrix_multiply_impl<unsigned long long>(0, 0, 0, nullptr, nullptr, nullptr);
		}
	}
}