#include <cstddef>
#include <utility>
#include <cmath>

#include <hip/hip_runtime.h>

namespace cmoon
{
	namespace cuda
	{
		template<class T>
		__global__
		void plus_ranges_kernel(int n, const T* const input1, const T* const input2, T* out)
		{
			const auto index = blockIdx.x * blockDim.x + threadIdx.x;
			const auto stride = blockDim.x * gridDim.x;
			for (auto i = index; i < n; i += stride)
			{
				out[i] = input1[i] + input2[i];
			}
		}

		template<class T>
		void plus_ranges_impl(int n, const T* const input1, const T* const input2, T* out)
		{
			constexpr auto blockSize = 256;
			const auto numBlocks = (n + blockSize - 1) / blockSize;
			plus_ranges_kernel<<<numBlocks, blockSize>>>(n, input1, input2, out);
		}

		namespace details
		{
			void dummy_template_instantiator()
			{
				plus_ranges_impl<float>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<double>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<char>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<unsigned char>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<short>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<unsigned short>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<int>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<unsigned int>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<long>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<unsigned long>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<long long>(0, nullptr, nullptr, nullptr);
				plus_ranges_impl<unsigned long long>(0, nullptr, nullptr, nullptr);
			}
		}
	}
}